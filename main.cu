#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <SFML/Audio.hpp>
#include <iostream>

const unsigned int WIDTH = 800;
const unsigned int HEIGHT = 600;
const float DEPTH = -700.0f; // for sound
const std::string WINDOW_TITLE = "Moon, Earth, Sun, Spaghetti Way!";

int main(int argc, char** argv[])
{
	sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), WINDOW_TITLE);
	window.setFramerateLimit(60);

	// Load background
	sf::Texture spaceTexture;
	spaceTexture.loadFromFile("stars.png");
	sf::Sprite background(spaceTexture);
	float textureWidth = spaceTexture.getSize().x;
	float textureHeight = spaceTexture.getSize().y;
	background.setScale(WIDTH / textureWidth, HEIGHT / textureHeight);

	// Load Sun
	sf::Texture sunTexture;
	sunTexture.loadFromFile("sun2.png");
	sf::Sprite sun(sunTexture);
	sun.setOrigin(sunTexture.getSize().x / 2, sunTexture.getSize().y / 2);
	sun.setScale(0.4f, 0.4f);
	sun.setPosition(WIDTH / 2, HEIGHT / 2);

	// Load Earth
	sf::Texture earthTexture;
	earthTexture.loadFromFile("earth3.png");
	sf::Sprite earth(earthTexture);
	earth.setOrigin(earthTexture.getSize().x / 2, earthTexture.getSize().y / 2);
	earth.setScale(0.1f, 0.1f);

	// Load Moon
	sf::Texture moonTexture;
	moonTexture.loadFromFile("moon1.png");
	sf::Sprite moon(moonTexture);
	moon.setOrigin(moonTexture.getSize().x / 2, moonTexture.getSize().y / 2);
	moon.setScale(0.01f, 0.01f);

	// Load earth sound
	sf::SoundBuffer buffer;
	buffer.loadFromFile("earthsound.wav");
	sf::Sound sound(buffer);
	sound.setPosition(WIDTH/2, HEIGHT/2, DEPTH/2);
	sound.play();
	sound.setLoop(true);
	sound.setRelativeToListener(false);
	sound.setMinDistance(50.0f);
	sound.setAttenuation(2.0);
	sf::Vector3f soundPosition(earth.getPosition().x, earth.getPosition().y, DEPTH / 2);

	// Load listener
	sf::Listener::setPosition(WIDTH/2, HEIGHT/2, -50.0f);
	sf::Listener::setDirection(0.0f, 0.0f, -1.0f);
	sf::Listener::setGlobalVolume(100.0f);

	float earthAngle = 3.0f;
	float moonAngle = 3.0f;

	while (window.isOpen())
	{
		sf::Event event;
		while (window.pollEvent(event))
		{
			if (event.type == sf::Event::Closed)
			{
				window.close();
			}
			if (event.key.code == sf::Keyboard::F11)
			{
				window.create(sf::VideoMode(WIDTH, HEIGHT),
					WINDOW_TITLE, sf::Style::Fullscreen);
				window.setFramerateLimit(60);
			}
			if (event.key.code == sf::Keyboard::Escape)
			{
				window.create(sf::VideoMode(WIDTH, HEIGHT),
					WINDOW_TITLE);
				window.setFramerateLimit(60);
			}
		}

		// Update angles of Earth and moon
		earthAngle += 0.004f;
		moonAngle -= 0.019f;
		// Update Earth position
		sf::Vector2f earthDelta(300 * cos(earthAngle), 100 * sin(earthAngle));
		earth.setPosition(sun.getPosition() + earthDelta);
		soundPosition.x = earth.getPosition().x;
		soundPosition.y = earth.getPosition().y;
		// Update Moon position
		float moonDistance = earth.getScale().x*500;
		sf::Vector2f moonDelta(moonDistance * cos(moonAngle), moonDistance * sin(moonAngle));
		moon.setPosition(earth.getPosition() + moonDelta);
		// Update Earth and moon sizes
		float fractionDown = 0.9990f;
		float fractionUp = 1.0010f;
		float fractionSound = 0.5000f;
		if (earth.getPosition().x > sun.getPosition().x)
		{
			earth.scale(fractionUp, fractionUp);
			moon.scale(fractionUp, fractionUp);
			soundPosition.z += fractionSound;
		}
		if (soundPosition.z < DEPTH)
			soundPosition.z = DEPTH;
		else if (earth.getPosition().x < sun.getPosition().x)
		{
			earth.scale(fractionDown, fractionDown);
			moon.scale(fractionDown, fractionDown);
			soundPosition.z -= fractionSound;
		}
		if (soundPosition.z > 0.0f)
			soundPosition.z = 0.0f;		
		sound.setPosition(soundPosition);

		window.clear();
		window.draw(background);

		if (earth.getPosition().y > sun.getPosition().y) // earth is close to view
		{
			window.draw(sun);
			window.draw(earth);
			window.draw(moon);
		}
		else
		{
			window.draw(earth);
			window.draw(moon);
			window.draw(sun);
		}

		window.display();
	}
}
