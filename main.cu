#include "hip/hip_runtime.h"
#include <stdio.h>

#define TILE_DIM 16

__global__
void matrixMultiply(float * A, float * B, float * C, int m, int n, int k)
{
    //Create 2 tiles for matrix A and B at the shared memory
    __shared__ float ATile[TILE_DIM][TILE_DIM];
    __shared__ float BTile[TILE_DIM][TILE_DIM];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int thrX = threadIdx.x;
    int thrY = threadIdx.y;

    //to accumulate partial values of each element in C
    float elementC = 0;

    for (int t = 0; t < (n-1)/TILE_DIM +1; ++t)
    {
        //threads to load matrix A to shared memory
        if(row < m && t*TILE_DIM+thrX < n)
            ATile[thrY][thrX] = A[row*n + t*TILE_DIM+thrX];
        else
            ATile[thrY][thrX] = 0.0f;

        //threads to load matrix B to shared memory
        if (t*TILE_DIM+thrY < n && col < k)
            BTile[thrY][thrX] = B[(t*TILE_DIM+thrY)*k + col];
        else
            BTile[thrY][thrX] = 0.0f;

        __syncthreads();

        //calculate a partial value of thread element in C
        for (int i = 0; i < TILE_DIM; ++i)
            elementC += ATile[thrY][i] * BTile[i][thrX];

        __syncthreads();

    }
    //copy final element value to the C matrix
    if (row < m && col < k)
        C[row*k+col] = elementC;

}

int main(int argc, char ** argv)
{
    float *hostA;
    float *hostB;
    float *hostC;

    float *deviceA;
    float *deviceB;
    float *deviceC;

    int m; // number of A rows
    int n; // number of A columns (or B rows)
    int k; // number of B columns

    printf("Enter m n and k\n");
    scanf("%d%d%d", &m, &n, &k);

    //allocate data in host
    hostA = (float *) malloc(m * n * sizeof(float));
    hostB = (float *) malloc(n * k * sizeof(float));
    hostC = (float *) malloc(m * k * sizeof(float));

    printf("Enter matrix A\n");
    for(int i = 0; i < m; i++)
        for(int j = 0; j < n; j++)
            scanf("%f", &hostA[i*m + j]);
    printf("Enter matrix B\n");
    for(int i = 0; i < n; i++)
        for(int j = 0; j < k; j++)
            scanf("&f", &hostB[i*n + j]);

    //allocate data in device
    hipMalloc((void **) &deviceA, m * n * sizeof(float));
    hipMalloc((void **) &deviceB, n * k * sizeof(float));
    hipMalloc((void **) &deviceC, m * k * sizeof(float));

    //copy inputs to device
    hipMemcpy(deviceA, hostA, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, n * k * sizeof(float), hipMemcpyHostToDevice);

    //device kernal
    dim3 DimGrid((k-1)/TILE_DIM+1, (m-1)/TILE_DIM+1, 1);
    dim3 DimBlock(TILE_DIM, TILE_DIM, 1);
    matrixMultiply<<<DimGrid,DimBlock>>>(deviceA, deviceB, deviceC, m, n, k);
    hipDeviceSynchronize();

    //copy result back to host
    hipMemcpy(hostC, deviceC, m * k * sizeof(float), hipMemcpyDeviceToHost);

    //deallocate device
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    printf("Matrix C\n");
    for(int i = 0; i < m; i++)
    {
        for(int j = 0; j < k; j++)
            scanf("&f ", &hostB[i*m + j]);
        printf("\n");
    }

    //deallocate host
    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
